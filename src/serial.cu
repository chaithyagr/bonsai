#include "hip/hip_runtime.h"
#include "types.h"
#include "buildtree.h"
#include "dataset.h"
#include "grouptargets.h"
#include "traversal.h"
#include "upwardpass.h"
#include "serial.hpp"

template < class T>
void print_random_data(const char *string, T pos, int num, int n)
  {
      int idx = 0;
      for(int i=0;i<num;++i)
      {
          idx = rand() % n;
          printf("%s : Data %d %d : %lf %lf %lf %lf\n",string,i, idx, pos[idx][0], pos[idx][1], pos[idx][2], pos[idx][3]);
      }
  }

template < class T>
void print_upto_n(const char * string, T pos, int num)
{
    for(int i = 0 ; i<num;++i)
        printf("%s : Body %d : %f\n", string, i, pos[i][0]);
}

int get_pots(int numBodies, float theta, float *pots, float *points, float *weights, bool dodebug=0)
{
    const int images = 0;
    const float eps = 0.05;
    const int ncrit = 32;
    const float cycle = 2 * M_PI;
    const Dataset data(numBodies, points, weights);

    fprintf(stdout,"--- FMM Parameters ---------------\n");
    fprintf(stdout,"numBodies            : %d\n",numBodies);
    fprintf(stdout,"P                    : %d\n",P);
    fprintf(stdout,"theta                : %f\n",theta);
    fprintf(stdout,"ncrit                : %d\n",ncrit);

    cudaVec<fvec4> bodyPos(numBodies,true);
    cudaVec<fvec4> bodyPos2(numBodies);
    cudaVec<fvec4> bodyAcc(numBodies,true);
    cudaVec<fvec4> bodyAcc2(numBodies,true);
    for (int i=0; i<numBodies; i++) {
      bodyPos[i][0] = data.pos[i][0];
      bodyPos[i][1] = data.pos[i][1];
      bodyPos[i][2] = data.pos[i][2];
      bodyPos[i][3] = data.pos[i][3];
    }
    bodyPos.h2d();
    bodyAcc.h2d();

    fprintf(stdout,"--- FMM Profiling ----------------\n");
    double t0 = get_time();
    Build build;
    Box box;
    cudaVec<int2> levelRange(32,true);
    cudaVec<CellData> sourceCells(numBodies);
    int3 counts = build.tree<ncrit>(bodyPos, bodyPos2, box, levelRange, sourceCells);
    int numLevels = counts.x;
    int numSources = counts.y;
    int numLeafs = counts.z;
    cudaVec<int2> targetRange(numBodies);
    cudaVec<fvec4> sourceCenter(numSources);
    cudaVec<fvec4> Multipole(NVEC4*numSources);
    Group group;
    int numTargets = group.targets(bodyPos, bodyPos2, box, targetRange, 5);
    Pass pass;
    pass.upward(numLeafs, numLevels, theta, levelRange, bodyPos, sourceCells, sourceCenter, Multipole);
    Traversal traversal;
    const fvec4 interactions = traversal.approx(numTargets, images, eps, cycle,
					      bodyPos, bodyPos2, bodyAcc,
					      targetRange, sourceCells, sourceCenter,
					      Multipole, levelRange);
    double dt = get_time() - t0;
    float flops = (interactions[0] * 20 + interactions[2] * 2 * pow(P,3)) * numBodies / dt / 1e12;
    fprintf(stdout,"--- Total runtime ----------------\n");
    fprintf(stdout,"Total FMM            : %.7f s (%.7f TFlops)\n",dt,flops);
    bodyAcc.d2h();
    for(int i=0;i<numBodies;++i)
        pots[i] = bodyAcc[i][0];
    fprintf(stdout,"Bodies               : %d\n",numBodies);
    fprintf(stdout,"Cells                : %d\n",numSources);
    fprintf(stdout,"Tree depth           : %d\n",numLevels);
    return 0;
}